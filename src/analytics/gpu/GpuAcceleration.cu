#include "hip/hip_runtime.h"
#include "GpuAcceleration.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vector_add(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

namespace gpu {

void parallel_sum(const std::vector<float>& a, const std::vector<float>& b, std::vector<float>& c) {
    if (a.size() != b.size()) {
        throw std::invalid_argument("Vectors must have the same size");
    }
    c.resize(a.size());

    float *d_a, *d_b, *d_c;
    size_t size = a.size() * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (a.size() + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, a.size());

    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

} // namespace gpu
