#include "hip/hip_runtime.h"
#include "analytics/gpu/GpuAcceleration.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

// CUDA Kernel for element-wise vector addition
__global__ void add_vectors_kernel(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

namespace gpu {

void parallel_sum(const std::vector<float>& a, const std::vector<float>& b, std::vector<float>& c) {
    if (a.size() != b.size()) {
        throw std::invalid_argument("Input vectors must have the same size.");
    }
    if (a.empty()) {
        c.clear();
        return;
    }

    const int n = a.size();
    c.resize(n);

    float *d_a, *d_b, *d_c;
    size_t size = n * sizeof(float);

    // Allocate memory on the GPU
    hipError_t err = hipMalloc((void**)&d_a, size);
    if (err != hipSuccess) throw std::runtime_error("Failed to allocate device memory for a.");
    
    err = hipMalloc((void**)&d_b, size);
    if (err != hipSuccess) {
        hipFree(d_a);
        throw std::runtime_error("Failed to allocate device memory for b.");
    }

    err = hipMalloc((void**)&d_c, size);
    if (err != hipSuccess) {
        hipFree(d_a);
        hipFree(d_b);
        throw std::runtime_error("Failed to allocate device memory for c.");
    }

    // Copy data from host to device
    err = hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("Failed to copy a to device.");
    }

    err = hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("Failed to copy b to device.");
    }

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    add_vectors_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    // Copy result from device to host
    err = hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("Failed to copy result from device.");
    }

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

} // namespace gpu
